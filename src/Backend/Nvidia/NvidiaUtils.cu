// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <iostream>
#include <map>
#include <vector>

#include "Nvidia/Argon2.h"
#include "Utilities/ColouredMsg.h"

#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>
#include <sstream>

int getDeviceCount()
{
    int numberDevices;

    bool haveDevice = throw_on_cuda_error(hipGetDeviceCount(&numberDevices), __FILE__, __LINE__);

    if (!haveDevice)
    {
        return 0;
    }

    return numberDevices;
}

std::string getDeviceName(uint16_t deviceId)
{
    hipDeviceProp_t prop;
    throw_on_cuda_error(hipGetDeviceProperties(&prop, deviceId), __FILE__, __LINE__);
    return prop.name;
}

std::vector<std::tuple<std::string, bool, int>> getNvidiaDevicesActual()
{
    std::vector<std::tuple<std::string, bool, int>> devices;

    int numberDevices = getDeviceCount();

    for (int i = 0; i < numberDevices; i++)
    {
        const auto device = getDeviceName(i);

        if (device == "")
        {
            continue;
        }

        devices.push_back(std::make_tuple(device, true, i));
    }

    return devices;
}

void printNvidiaHeader()
{
    std::cout << InformationMsg<std::string>("* ") << WhiteMsg<std::string>("NVIDIA DEVICES", 25);

    int numberDevices = getDeviceCount();

    if (numberDevices == 0)
    {
        std::cout << WarningMsg<std::string>("None found") << std::endl;
        return;
    }

    std::map<std::string, std::vector<hipDeviceProp_t>> gpus;

    for (int i = 0; i < numberDevices; i++)
    {
        hipDeviceProp_t prop;
        throw_on_cuda_error(hipGetDeviceProperties(&prop, i), __FILE__, __LINE__);

        std::string deviceName = prop.name;

        gpus[deviceName].push_back(prop);
    }

    int i = 0;

    for (const auto gpu : gpus)
    {
        const auto name = gpu.first;
        const auto properties = gpu.second;

        if (i != 0)
        {
            std::cout << ", ";
        }

        size_t numDevices = properties.size();

        std::cout << SuccessMsg<size_t>(numDevices) << SuccessMsg<std::string>("x ") << SuccessMsg<std::string>(name);

        i++;
    }

    std::cout << std::endl;
}
