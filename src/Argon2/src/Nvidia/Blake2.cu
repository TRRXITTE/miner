#include "hip/hip_runtime.h"
// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <cstring>
#include <stdint.h>
#include <string>
#include <iostream>
#include <vector>
#include <sstream>

#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>

#include "Blake2.h"
#include "Argon2.h"

#define IV0 0x6a09e667f3bcc908UL
#define IV1 0xbb67ae8584caa73bUL
#define IV2 0x3c6ef372fe94f82bUL
#define IV3 0xa54ff53a5f1d36f1UL
#define IV4 0x510e527fade682d1UL
#define IV5 0x9b05688c2b3e6c1fUL
#define IV6 0x1f83d9abfb41bd6bUL
#define IV7 0x5be0cd19137e2179UL

__constant__ static const uint8_t sigma[12][16] =
{
    { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 },
    { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  },
    { 11, 8,  12, 0,  5,  2,  15, 13, 10, 14, 3,  6,  7,  1,  9,  4  },
    { 7,  9,  3,  1,  13, 12, 11, 14, 2,  6,  5,  10, 4,  0,  15, 8  },
    { 9,  0,  5,  7,  2,  4,  10, 15, 14, 1,  11, 12, 6,  8,  3,  13 },
    { 2,  12, 6,  10, 0,  11, 8,  3,  4,  13, 7,  5,  15, 14, 1,  9  },
    { 12, 5,  1,  15, 14, 13, 4,  10, 0,  7,  6,  3,  9,  2,  8,  11 },
    { 13, 11, 7,  14, 12, 1,  3,  9,  5,  0,  15, 4,  8,  6,  2,  10 },
    { 6,  15, 14, 9,  11, 3,  0,  8,  12, 2,  13, 7,  1,  4,  10, 5  },
    { 10, 2,  8,  4,  7,  6,  1,  5,  15, 11, 9,  14, 3,  12, 13, 0  },
    { 0,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15 },
    { 14, 10, 4,  8,  9,  15, 13, 6,  1,  12, 0,  2,  11, 7,  5,  3  },
};

__device__ __forceinline__
uint64_t rotr64(uint64_t x, uint32_t n)
{
    return (x >> n) | (x << (64 - n));
}

__device__
void blake2b_init(uint64_t *h, uint32_t hashlen)
{
    h[0] = IV0 ^ (0x01010000 | hashlen);
    h[1] = IV1;
    h[2] = IV2;
    h[3] = IV3;
    h[4] = IV4;
    h[5] = IV5;
    h[6] = IV6;
    h[7] = IV7;
}

__device__ void g(uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, uint64_t m1, uint64_t m2)
{
    asm("{"
        ".reg .u64 s, x;"
        ".reg .u32 l1, l2, h1, h2;"
        // a = a + b + x
        "add.u64 %0, %0, %1;"
        "add.u64 %0, %0, %4;"
        // d = rotr64(d ^ a, 32)
        "xor.b64 x, %3, %0;"
        "mov.b64 {h1, l1}, x;"
        "mov.b64 %3, {l1, h1};"
        // c = c + d
        "add.u64 %2, %2, %3;"
        // b = rotr64(b ^ c, 24)
        "xor.b64 x, %1, %2;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x6543;"
        "prmt.b32 h2, l1, h1, 0x2107;"
        "mov.b64 %1, {l2, h2};"
        // a = a + b + y
        "add.u64 %0, %0, %1;"
        "add.u64 %0, %0, %5;"
        // d = rotr64(d ^ a, 16);
        "xor.b64 x, %3, %0;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x5432;"
        "prmt.b32 h2, l1, h1, 0x1076;"
        "mov.b64 %3, {l2, h2};"
        // c = c + d
        "add.u64 %2, %2, %3;"
        // b = rotr64(b ^ c, 63)
        "xor.b64 x, %1, %2;"
        "shl.b64 s, x, 1;"
        "shr.b64 x, x, 63;"
        "add.u64 %1, s, x;"
        "}"
        : "+l"(*a), "+l"(*b), "+l"(*c), "+l"(*d) : "l"(m1), "l"(m2)
    );
}

#define G(i, a, b, c, d) (g(&v[a], &v[b], &v[c], &v[d], m[sigma[r][2 * i]], m[sigma[r][2 * i + 1]]))

__device__ void blake2b_round(uint32_t r, uint64_t *v, uint64_t *m)
{
    G(0, 0, 4, 8, 12);
    G(1, 1, 5, 9, 13);
    G(2, 2, 6, 10, 14);
    G(3, 3, 7, 11, 15);
    G(4, 0, 5, 10, 15);
    G(5, 1, 6, 11, 12);
    G(6, 2, 7, 8, 13);
    G(7, 3, 4, 9, 14);
}

__device__ void blake2b_compress(
    uint64_t *h,
    uint64_t *m,
    uint32_t bytes_compressed,
    const bool last_block)
{
    uint64_t v[ARGON_QWORDS_IN_BLOCK];

    v[0] = h[0];
    v[1] = h[1];
    v[2] = h[2];
    v[3] = h[3];
    v[4] = h[4];
    v[5] = h[5];
    v[6] = h[6];
    v[7] = h[7];
    v[8] = IV0;
    v[9] = IV1;
    v[10] = IV2;
    v[11] = IV3;
    v[12] = IV4 ^ bytes_compressed;
    v[13] = IV5; // it's OK if below 2^32 bytes
    v[14] = last_block ? ~IV6 : IV6;
    v[15] = IV7;

    #pragma unroll
    for (uint32_t r = 0; r < 12; r++)
    {
        blake2b_round(r, v, m);
    }

    h[0] = h[0] ^ v[0] ^ v[8];
    h[1] = h[1] ^ v[1] ^ v[9];
    h[2] = h[2] ^ v[2] ^ v[10];
    h[3] = h[3] ^ v[3] ^ v[11];
    h[4] = h[4] ^ v[4] ^ v[12];
    h[5] = h[5] ^ v[5] ^ v[13];
    h[6] = h[6] ^ v[6] ^ v[14];
    h[7] = h[7] ^ v[7] ^ v[15];
}

__device__ __forceinline__
void setNonce(
    uint64_t *inseed,
    uint32_t nonce,
    const uint64_t nonceMask)
{
    /* Need 64 bit to do a shift of 40 */
    uint64_t nonce64 = nonce;

    /* Set byte 68-70 or 67-70 depending on whether this is a nicehash job or not */
    inseed[8] = inseed[8] | ((nonce64 << 24) & nonceMask);
}

__device__
void initial_hash(
    uint64_t *hash,
    uint64_t *inseed,
    size_t blakeInputSize,
    uint32_t nonce,
    const uint64_t nonceMask)
{
    uint64_t buffer[BLAKE_QWORDS_IN_BLOCK];

    blake2b_init(hash, BLAKE_HASH_LENGTH);

    for (int i = 0; i < BLAKE_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = inseed[i];
    }

    setNonce(buffer, nonce, nonceMask);

    blake2b_compress(hash, buffer, BLAKE_BLOCK_SIZE, false);

    for (int i = 0; i < BLAKE_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = inseed[BLAKE_QWORDS_IN_BLOCK + i];
    }

    blake2b_compress(hash, buffer, blakeInputSize, true);
}

__device__
void fillFirstBlock(
    block_g *memory,
    uint64_t *blakeInput,
    size_t blakeInputSize,
    uint32_t nonce,
    uint32_t block,
    const uint64_t nonceMask)
{
    uint64_t hash[8];
    initial_hash(hash, blakeInput, blakeInputSize, nonce, nonceMask);

    uint32_t prehash_seed[BLAKE_DWORDS_IN_BLOCK];

    prehash_seed[0] = ARGON_BLOCK_SIZE;

    memcpy(&prehash_seed[1], hash, BLAKE_HASH_LENGTH);

    prehash_seed[17] = block;

    for (int i = 18; i < BLAKE_DWORDS_IN_BLOCK; i++)
    {
        prehash_seed[i] = 0;
    }

    uint64_t *dst = static_cast<uint64_t *>(memory->data);

    blake2b_init(hash, BLAKE_HASH_LENGTH);
    blake2b_compress(hash, reinterpret_cast<uint64_t *>(prehash_seed), BLAKE_INITIAL_HASH_LENGTH, true);

    *(dst++) = hash[0];
    *(dst++) = hash[1];
    *(dst++) = hash[2];
    *(dst++) = hash[3];

    uint64_t buffer[BLAKE_QWORDS_IN_BLOCK];

    for (int i = 8; i < BLAKE_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = 0;
    }

    for (int r = 2; r < 2 * ARGON_BLOCK_SIZE / BLAKE_HASH_LENGTH; r++)
    {
        buffer[0] = hash[0];
        buffer[1] = hash[1];
        buffer[2] = hash[2];
        buffer[3] = hash[3];
        buffer[4] = hash[4];
        buffer[5] = hash[5];
        buffer[6] = hash[6];
        buffer[7] = hash[7];

        blake2b_init(hash, BLAKE_HASH_LENGTH);
        blake2b_compress(hash, buffer, BLAKE_HASH_LENGTH, true);

        *(dst++) = hash[0];
        *(dst++) = hash[1];
        *(dst++) = hash[2];
        *(dst++) = hash[3];
    }

    *(dst++) = hash[4];
    *(dst++) = hash[5];
    *(dst++) = hash[6];
    *(dst++) = hash[7];
}

__device__
void hash_last_block(block_g *memory, uint64_t *hash)
{
    uint64_t buffer[BLAKE_QWORDS_IN_BLOCK];
    uint32_t hi, lo;
    uint32_t bytes_compressed = 0;
    uint32_t bytes_remaining = ARGON_BLOCK_SIZE;

    uint32_t *src = reinterpret_cast<uint32_t *>(memory->data);

    blake2b_init(hash, ARGON_HASH_LENGTH);

    hi = *(src++);
    buffer[0] = 32 | ((uint64_t)hi << 32);

    #pragma unroll
    for (uint32_t i = 1; i < BLAKE_QWORDS_IN_BLOCK; i++)
    {
        lo = *(src++);
        hi = *(src++);
        buffer[i] = lo | ((uint64_t)hi << 32);
    }

    bytes_compressed += BLAKE_BLOCK_SIZE;
    bytes_remaining -= (BLAKE_BLOCK_SIZE - sizeof(uint32_t));
    blake2b_compress(hash, buffer, bytes_compressed, false);

    while (bytes_remaining > BLAKE_BLOCK_SIZE)
    {
        #pragma unroll
        for (uint32_t i = 0; i < BLAKE_QWORDS_IN_BLOCK; i++)
        {
            lo = *(src++);
            hi = *(src++);
            buffer[i] = lo | ((uint64_t)hi << 32);
        }

        bytes_compressed += BLAKE_BLOCK_SIZE;
        bytes_remaining -= BLAKE_BLOCK_SIZE;
        blake2b_compress(hash, buffer, bytes_compressed, false);
    }

    buffer[0] = *src;

    #pragma unroll
    for (uint32_t i = 1; i < BLAKE_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = 0;
    }

    bytes_compressed += bytes_remaining;
    blake2b_compress(hash, buffer, bytes_compressed, true);
}

__global__
void initMemoryKernel(
    block_g *memory,
    uint64_t *blakeInput,
    size_t blakeInputSize,
    const uint32_t startNonce,
    const size_t scratchpadSize,
    const uint64_t nonceMask)
{
    uint32_t jobNumber = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = startNonce + jobNumber;
    uint32_t block = threadIdx.y;

    /* Find the index for the memory belonging to this GPU thread */
    block_g *threadMemory = memory + (static_cast<uint64_t>(jobNumber) * scratchpadSize + block);

    fillFirstBlock(threadMemory, blakeInput, blakeInputSize, nonce, block, nonceMask);
}

__global__
void getNonceKernel(
    block_g *memory,
    const uint32_t startNonce,
    uint64_t target,
    uint32_t *resultNonce,
    uint8_t *resultHash,
    bool *success,
    const size_t scratchpadSize,
    const bool isNiceHash,
    const uint64_t *blakeInput)
{
    uint32_t jobNumber = blockIdx.x * blockDim.x + threadIdx.x;

    block_g *threadMemory = memory + (static_cast<uint64_t>(jobNumber) + 1) * scratchpadSize - 1;

    uint64_t hash[8];

    hash_last_block(threadMemory, hash);

    /* Valid hash, notify success and copy hash */
    if (hash[3] < target)
    {
        uint32_t storedNonce = static_cast<uint32_t>(blakeInput[8] >> 24);

        uint32_t nonce = startNonce + jobNumber;

        if (isNiceHash)
        {
            nonce = (nonce & 0x00FFFFFF) | (storedNonce & 0xFF000000);
        }

        /* Store the successful nonce in resultNonce if it's currently set
           to zero. */
        uint32_t old = atomicCAS(resultNonce, 0, nonce);

        /* If the returned value is zero, then this is the first thread to
           find a nonce. Lets store the corresponding hash. */
        if (old == 0)
        {
            *success = true;

            #pragma unroll
            for (int i = 0; i < 4; i++)
            {
                *reinterpret_cast<uint64_t *>(resultHash + (i * 8)) = hash[i];
            }
        }
    }
}

void setupBlakeInput(
    const std::vector<uint8_t> &input,
    const std::vector<uint8_t> &saltInput,
    NvidiaState &state)
{
    const uint32_t threads = 1;
    const uint32_t keyLen = ARGON_HASH_LENGTH;
    const uint32_t memory = state.launchParams.scratchpadSize;
    const uint32_t time = state.launchParams.iterations;
    const uint32_t version = 19; /* Argon version */
    const uint32_t mode = 2; /* Argon2id */

    const uint32_t messageSize = static_cast<uint32_t>(input.size());
    const uint32_t saltSize = saltInput.size();
    const uint32_t secretSize = 0;
    const uint32_t dataSize = 0;

    state.blakeInputSize = sizeof(threads) + sizeof(keyLen) + sizeof(memory)
        + sizeof(time) + sizeof(version) + sizeof(mode) + sizeof(messageSize)
        + messageSize + sizeof(saltSize) + saltSize + sizeof(secretSize) + secretSize
        + sizeof(dataSize) + dataSize;

    /* We pad the data to BLOCK_SIZE * 2. Max block header is supposedly 128 bytes. */
    uint8_t initialInput[BLAKE_BLOCK_SIZE * 2] = {};

    size_t index = 0;

    std::memcpy(&initialInput[index], &threads, sizeof(threads));
    index += sizeof(threads);

    std::memcpy(&initialInput[index], &keyLen, sizeof(keyLen));
    index += sizeof(keyLen);

    std::memcpy(&initialInput[index], &memory, sizeof(memory));
    index += sizeof(memory);

    std::memcpy(&initialInput[index], &time, sizeof(time));
    index += sizeof(time);

    std::memcpy(&initialInput[index], &version, sizeof(version));
    index += sizeof(version);

    std::memcpy(&initialInput[index], &mode, sizeof(mode));
    index += sizeof(mode);

    std::memcpy(&initialInput[index], &messageSize, sizeof(messageSize));
    index += sizeof(messageSize);

    std::memcpy(&initialInput[index], &input[0], messageSize);
    index += messageSize;

    std::memcpy(&initialInput[index], &saltSize, sizeof(saltSize));
    index += sizeof(saltSize);

    std::memcpy(&initialInput[index], &saltInput[0], saltSize);
    index += saltSize;

    std::memcpy(&initialInput[index], &secretSize, sizeof(secretSize));
    index += sizeof(secretSize);

    std::memcpy(&initialInput[index], &dataSize, sizeof(dataSize));
    index += sizeof(dataSize);

    /* Copy over the input data */
    throw_on_cuda_error(hipMemcpyAsync(state.blakeInput, &initialInput[0], BLAKE_BLOCK_SIZE * 2, hipMemcpyHostToDevice, state.stream), __FILE__, __LINE__);
}
